#include "hip/hip_runtime.h"
%%writefile matrix_mul_1_0.cu
#include <iostream>
#include <hip/hip_runtime.h>

#define ERR_CHECK(err)                                                                                                  \
    if (err != hipSuccess)                                                                                             \
    {                                                                                                                   \
        std::cout << "Error with cuda operation : " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        return;                                                                                                         \
    }

        __global__ void
        multiplyVector2D(float *A, size_t pitch_A, float *B, size_t pitch_B, float *C, size_t pitch_C, int width_A, int height_A, int width_B, int height_B, int width_C, int height_C)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    float *r_A = (float *)((char *)A + row * pitch_A);
    float *r_C = (float *)((char *)C + row * pitch_C);

    if (col < width_C && row < height_C)
    {
        float temp = 0.0f;
        for (int i = 0; i < width_A; i++)
        {
            float *r_B = (float *)((char *)B + i * pitch_B);
            temp += r_A[i] * r_B[col];
        }
        r_C[col] = temp;
    }

    return;
}
void deviceConfig()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
    std::cout << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    std::cout << "  Max block dimensions (X, Y, Z): ("
              << deviceProp.maxThreadsDim[0] << ", "
              << deviceProp.maxThreadsDim[1] << ", "
              << deviceProp.maxThreadsDim[2] << ")" << std::endl;
    std::cout << std::endl;
}

void twoDOperations()
{
    size_t width_A = 1 << 12;
    size_t height_A = 1 << 12;
    size_t width_B = 1 << 12;
    size_t height_B = 1 << 12;
    size_t width_C = height_A;
    size_t height_C = width_B;
    // int N = width * height;
    // size_t size = N * sizeof(float);

    float *h_A = new float[height_A * width_A];
    float *h_B = new float[height_B * width_B];
    float *h_C = new float[height_C * width_C];

    deviceConfig();

    // generate data

    for (int i = 0; i < height_A; i++)
    {
        for (int j = 0; j < width_A; j++)
        {
            h_A[i * width_A + j] = (1);
        }
    }

    for (int i = 0; i < height_B; i++)
    {
        for (int j = 0; j < width_B; j++)
        {
            h_B[i * width_B + j] = (j);
        }
    }

    hipSetDevice(0);
    ERR_CHECK(hipGetLastError());

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipStream_t stream_2;
    hipStreamCreate(&stream_2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *d_A, *d_B, *d_C;
    size_t pitch_A, pitch_B, pitch_C;

    hipMallocPitch((void **)&d_A, &pitch_A, (sizeof(float) * width_A), height_A);
    hipMallocPitch((void **)&d_B, &pitch_B, (sizeof(float) * width_B), height_B);
    hipMallocPitch((void **)&d_C, &pitch_C, (sizeof(float) * width_C), height_C);

    hipEventRecord(start, 0);
    hipMemcpy2DAsync(d_A, pitch_A, h_A, width_A * sizeof(float), width_A * sizeof(float), height_A, hipMemcpyHostToDevice, stream);
    hipMemcpy2DAsync(d_B, pitch_B, h_B, width_B * sizeof(float), width_B * sizeof(float), height_B, hipMemcpyHostToDevice, stream_2);

    // int threadsPerBlock = 256;
    // int numBlocks = width / threadsPerBlock;

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width_C + 15) / 16, (height_C + 15) / 16);
    // hipDeviceSynchronize();

    multiplyVector2D<<<numBlocks, threadsPerBlock, 0, stream>>>(d_A, pitch_A, d_B, pitch_B, d_C, pitch_C, width_A, height_A, width_B, height_B, width_C, height_C);
    hipEventRecord(stop, 0);
    ERR_CHECK(hipGetLastError());

    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Kernel took " << ms << " ms\n"
              << std::endl;

    hipMemcpy2D(h_C, width_C * sizeof(float), d_C, pitch_C, width_C * sizeof(float), height_C, hipMemcpyDeviceToHost);

    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            std::cout << "Index " << i << ", " << j << ", Value " << h_C[i * width_C + j] << std::endl;
        }
    }

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    twoDOperations();
    return 0;
}